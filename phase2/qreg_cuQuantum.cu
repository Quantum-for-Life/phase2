#include "hip/hip_runtime.h"
#include <complex.h>
#include <stddef.h>

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#include "custatevec.h"

#include "phase2/paulis.h"
#include "phase2/qreg.h"
#include "phase2/world.h"
#include "qreg_cuQuantum.h"
#include "world_cuQuantum.h"

const size_t threadPerBlock = 512;

__global__ void kernelAdd(hipDoubleComplex *a, hipDoubleComplex *b, size_t n)
{
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < n)
		a[i] = hipCadd(a[i], b[i]);

}

__global__ void kernelMul(hipDoubleComplex *a, hipDoubleComplex b, size_t n)
{
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < n)
		a[i] = hipCmul(a[i], b);
}

__global__ void kernelMix(hipDoubleComplex *a, hipDoubleComplex *b, size_t n)
{
	hipDoubleComplex z1, z2;
	const hipDoubleComplex half = { .x = 0.5, .y = 0.0 };

	size_t i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < n) {
		z1 = hipCadd(a[i], b[i]);
		z2 = hipCsub(a[i], b[i]);
		a[i] = hipCmul(z1, half);
		b[i] = hipCmul(z2, half);
	}
}



void qreg_paulirot_local(struct qreg *reg, custatevecHandle_t handle,
	       const struct paulis *codes_lo, const double *angles,
	       const size_t num_codes, double _Complex buf_mul)
{
 	const size_t blocks = (reg->num_amps + threadPerBlock - 1) / threadPerBlock;

	const struct qreg_cuQuantum *cu =
		(const struct qreg_cuQuantum *)reg->data;
	custatevecPauli_t paulis[QREG_MAX_WIDTH];

	hipMemcpy(cu->d_sv, reg->amp, sizeof(double) * 2 * reg->num_amps,
			hipMemcpyHostToDevice);
	hipMemcpy(cu->d_buf, reg->buf, sizeof(double) * 2 * reg->num_amps,
			hipMemcpyHostToDevice);

	/* Note that we're taking the conjugation of buf_mul. */
	const hipDoubleComplex b = { .x = creal(buf_mul), .y = -cimag(buf_mul) };
	kernelMul<<<blocks, threadPerBlock>>>(cu->d_buf, b, reg->num_amps);
	kernelMix<<<blocks, threadPerBlock>>>(cu->d_sv, cu->d_buf, reg->num_amps);

	for (size_t k = 0; k < num_codes; k++) {
		/* cuQuantum Paulis are the same as ours:
		 * CUSTATEVEC_PAULI_I = PAULI_I = 0, etc.
		 */
		for (size_t i = 0; i < cu->num_qubits; i++)
			paulis[i] = (custatevecPauli_t)paulis_get(codes_lo[k], i);

		// apply exponential
		custatevecApplyPauliRotation(
			handle, cu->d_sv, HIP_C_64F,
			cu->num_qubits, angles[k], paulis,
			cu->targs, cu->num_targs,
	    		nullptr, nullptr, 0);
		custatevecApplyPauliRotation(
			handle, cu->d_buf, HIP_C_64F,
			cu->num_qubits, -angles[k], paulis,
			cu->targs, cu->num_targs,
	    		nullptr, nullptr, 0);
	}

        kernelAdd<<<blocks, threadPerBlock>>>(cu->d_sv, cu->d_buf, reg->num_amps);

	hipMemcpy(reg->amp, cu->d_sv, sizeof(double) * 2 * reg->num_amps,
			hipMemcpyDeviceToHost);
	hipMemcpy(reg->buf, cu->d_buf, sizeof(double) * 2 * reg->num_amps,
			hipMemcpyDeviceToHost);

}
