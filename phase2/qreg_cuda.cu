#include "hip/hip_runtime.h"
#include <complex.h>
#include <stddef.h>

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#include "phase2/paulis.h"
#include "phase2/qreg.h"
#include "phase2/world.h"
#include "qreg_cuda.h"
#include "world_cuda.h"

const size_t threadPerBlock = 512;

__global__ void kernelAdd(hipDoubleComplex *a, hipDoubleComplex *b, size_t n)
{
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < n)
		a[i] = hipCadd(a[i], b[i]);

}

__global__ void kernelMul(hipDoubleComplex *a, hipDoubleComplex b, size_t n)
{
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < n)
		a[i] = hipCmul(a[i], b);
}

__global__ void kernelMix(hipDoubleComplex *a, hipDoubleComplex *b, size_t n)
{
	hipDoubleComplex z1, z2;
	const hipDoubleComplex half = { .x = 0.5, .y = 0.0 };

	size_t i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < n) {
		z1 = hipCadd(a[i], b[i]);
		z2 = hipCsub(a[i], b[i]);
		a[i] = hipCmul(z1, half);
		b[i] = hipCmul(z2, half);
	}
}

__global__ void kernelPauliRot(hipDoubleComplex *a, size_t n, hipDoubleComplex eip,
		struct paulis code)
{
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n)
		return;

	size_t j = i ^ code.pak[0];
	if (j < i)
		return;

	int minus = __popcll(j & code.pak[1]);
	int root4 = (__popcll(code.pak[0] & code.pak[1]) + 2*minus) & 0x3;
	hipDoubleComplex z;
	switch (root4) {
		case 0:
			z.x = 1.0;
			z.y = 0.0;
			break;
		case 1:
			z.x = 0.0;
			z.y = -1.0;
			break;
		case 2:
			z.x = -1.0;
			z.y = 0.0;
			break;
		case 3:
			z.x = 0.0;
			z.y = 1.0;
			break;
		default:
			__builtin_unreachable();
	}

	hipDoubleComplex zi, zj;
	zi = a[i];
	zj = a[j];

	hipDoubleComplex rc = { .x = hipCreal(eip), .y = 0.0 };
	hipDoubleComplex is = { .x = 0.0, .y = hipCimag(eip) };
	a[i] = hipCadd(hipCmul(rc, zi), hipCmul(is, hipCmul(hipConj(z), zj)));
	a[j] = hipCadd(hipCmul(rc, zj), hipCmul(is, hipCmul(z, zi)));
}

void qreg_paulirot_local(struct qreg *reg,
	       const struct paulis *codes_lo, const double *angles,
	       const size_t num_codes, double _Complex buf_mul)
{
 	const size_t blocks = (reg->num_amps + threadPerBlock - 1) / 
				threadPerBlock;

	const struct qreg_cuQuantum *cu =
		(const struct qreg_cuQuantum *)reg->data;

	/* Note that we're taking the conjugation of buf_mul. */
	const hipDoubleComplex b = { .x = creal(buf_mul), .y = -cimag(buf_mul) };
	kernelMul<<<blocks, threadPerBlock>>>(cu->d_buf, b, reg->num_amps);
	kernelMix<<<blocks, threadPerBlock>>>(cu->d_sv, cu->d_buf, reg->num_amps);

	hipDeviceSynchronize();
	for (size_t k = 0; k < num_codes; k++) {
		hipDoubleComplex eip = {
			.x = cos(angles[k]),
			.y = sin(angles[k])
		};
		kernelPauliRot<<<blocks, threadPerBlock>>>
			(cu->d_sv, reg->num_amps, eip, codes_lo[k]);
		kernelPauliRot<<<blocks, threadPerBlock>>>
			(cu->d_buf, reg->num_amps, hipConj(eip), codes_lo[k]);

	}

	/* We mix again d_sv and d_buf. Sync them first. */
	hipDeviceSynchronize();
        kernelAdd<<<blocks, threadPerBlock>>>(cu->d_sv, cu->d_buf,
			reg->num_amps);
}
