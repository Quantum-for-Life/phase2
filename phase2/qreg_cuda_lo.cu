#include "hip/hip_runtime.h"
#include <complex.h>
#include <stddef.h>

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>

#include "phase2/paulis.h"
#include "phase2/qreg.h"
#include "qreg_cuda.h"

constexpr size_t threadPerBlock = 512;
constexpr hipDoubleComplex half = { .x = 0.5, .y = 0.0 };

__global__ void kernelMix(
	hipDoubleComplex *a, hipDoubleComplex *b, hipDoubleComplex bm, size_t n)
{
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n)
		return;

	b[i] = hipCmul(b[i], bm);

	const hipDoubleComplex z1 = hipCadd(a[i], b[i]);
	const hipDoubleComplex z2 = hipCsub(a[i], b[i]);
	a[i] = hipCmul(z1, half);
	b[i] = hipCmul(z2, half);
}

__global__ void kernelPauliRot(
	hipDoubleComplex *a, size_t n, struct paulis code, double c, double s)
{
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n)
		return;

	size_t j = i ^ code.pak[0];
	if (j < i)
		return;

	hipDoubleComplex z = { .x = 1.0, .y = 0.0 };
	{
		int mi = __popcll(i & code.pak[1]); // no. of minuses
		int is = __popcll(code.pak[0] & code.pak[1]); // no. of i's
		int r4 = (is + 2 * mi) & 0x3; // 4th root of unity
		switch (r4) {
		case 0:
			break;
		case 1:
			z.x = 0.0;
			z.y = 1.0;
			break;
		case 2:
			z.x = -1.0;
			break;
		case 3:
			z.x = 0.0;
			z.y = -1.0;
			break;
		default:
			unreachable();
		}
	}

	const hipDoubleComplex zi = a[i];
	const hipDoubleComplex zj = a[j];
	const hipDoubleComplex rc = { .x = c, .y = 0.0 };
	const hipDoubleComplex is = { .x = 0.0, .y = s };

	a[i] = hipCadd(hipCmul(rc, zi), hipCmul(is, hipCmul(hipConj(z), zj)));
	a[j] = hipCadd(hipCmul(rc, zj), hipCmul(is, hipCmul(z, zi)));
}

__global__ void kernelAdd(hipDoubleComplex *a, hipDoubleComplex *b, size_t n)
{
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n)
		return;

	a[i] = hipCadd(a[i], b[i]);
}

void qreg_paulirot_lo(struct qreg *reg, const struct paulis *codes_lo,
	const double *angles, const size_t ncodes, double _Complex bm)
{
	const size_t blocks = (reg->namp + threadPerBlock - 1) / threadPerBlock;
	const struct qreg_cuda *cu = (const struct qreg_cuda *)reg->data;

	const hipDoubleComplex z = { .x = creal(bm), .y = cimag(bm) };
	kernelMix<<<blocks, threadPerBlock>>>(cu->damp, cu->dbuf, z, reg->namp);

	// hipDeviceSynchronize();
	for (size_t k = 0; k < ncodes; k++) {
		double c = cos(angles[k]), s = sin(angles[k]);
		kernelPauliRot<<<blocks, threadPerBlock>>>(
			cu->damp, reg->namp, codes_lo[k], c, s);
		kernelPauliRot<<<blocks, threadPerBlock>>>(
			cu->dbuf, reg->namp, codes_lo[k], c, -s);
	}

	// We mix again d_amp and d_buf. Sync them first.
	hipDeviceSynchronize();
	kernelAdd<<<blocks, threadPerBlock>>>(cu->damp, cu->dbuf, reg->namp);
}
