#include "hip/hip_runtime.h"
#include <complex.h>
#include <stddef.h>

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>

#include "phase2/paulis.h"
#include "phase2/qreg.h"
#include "qreg_cuda.h"

constexpr size_t threadPerBlock = 512;

constexpr hipDoubleComplex half = { .x = 0.5, .y = 0.0 };

__global__ void kernelMix(hipDoubleComplex *__restrict__ a,
	hipDoubleComplex *__restrict__ b, hipDoubleComplex bm, size_t n)
{
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n)
		return;

	b[i] = hipCmul(b[i], bm);

	const hipDoubleComplex z1 = hipCadd(a[i], b[i]);
	const hipDoubleComplex z2 = hipCsub(a[i], b[i]);
	a[i] = hipCmul(z1, half);
	b[i] = hipCmul(z2, half);
}

// Assume z != nullptr
__device__ uint64_t paulisEffect(
	const struct paulis code, const uint64_t i, hipDoubleComplex *z)
{
	int mi = __popcll(i & code.pak[1]); // no. of minuses
	int is = __popcll(code.pak[0] & code.pak[1]); // no. of i's
	int r4 = (is + 2 * mi) & 0x3; // 4th root of unity
	switch (r4) {
	case 0:
		break;
	case 1:
		*z = hipCmul(*z, (hipDoubleComplex){ .x = 0.0, .y = 1.0 });
		break;
	case 2:
		*z = hipCmul(*z, (hipDoubleComplex){ .x = -1.0, .y = 0.0 });
		break;
	case 3:
		*z = hipCmul(*z, (hipDoubleComplex){ .x = 0.0, .y = -1.0 });
		break;
	default:
		unreachable();
	}

	return i ^ code.pak[0];
}

__global__ void kernelPauliRot(
	hipDoubleComplex *a, size_t n, struct paulis code, double c, double s)
{
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n)
		return;

	hipDoubleComplex z = { .x = 1.0, .y = 0.0 };
	const uint64_t j = paulisEffect(code, i, &z);
	if (j < i)
		return;

	const hipDoubleComplex zi = a[i];
	const hipDoubleComplex zj = a[j];
	const hipDoubleComplex rc = { .x = c, .y = 0.0 };
	const hipDoubleComplex is = { .x = 0.0, .y = s };

	a[i] = hipCadd(hipCmul(rc, zi), hipCmul(is, hipCmul(hipConj(z), zj)));
	a[j] = hipCadd(hipCmul(rc, zj), hipCmul(is, hipCmul(z, zi)));
}

__global__ void kernelAdd(hipDoubleComplex *__restrict__ a,
	hipDoubleComplex *__restrict__ b, size_t n)
{
	size_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n)
		return;

	a[i] = hipCadd(a[i], b[i]);
}

void qreg_paulirot_lo(struct qreg *reg, const struct paulis *codes_lo,
	const double *angles, const size_t ncodes, double _Complex bm)
{
	const size_t blocks = (reg->namp + threadPerBlock - 1) / threadPerBlock;
	const struct qreg_cuda *cu = (const struct qreg_cuda *)reg->data;

	const hipDoubleComplex z = { .x = creal(bm), .y = cimag(bm) };
	kernelMix<<<blocks, threadPerBlock>>>(cu->damp, cu->dbuf, z, reg->namp);

	for (size_t k = 0; k < ncodes; k++) {
		double c = cos(angles[k]), s = sin(angles[k]);
		kernelPauliRot<<<blocks, threadPerBlock>>>(
			cu->damp, reg->namp, codes_lo[k], c, s);
		kernelPauliRot<<<blocks, threadPerBlock>>>(
			cu->dbuf, reg->namp, codes_lo[k], c, -s);
	}

	// We mix again damp and dbuf. Sync them first.
	hipDeviceSynchronize();
	kernelAdd<<<blocks, threadPerBlock>>>(cu->damp, cu->dbuf, reg->namp);
}
